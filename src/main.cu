#include "kernel.cuh"
#include "randomStatesKernel.cuh"
#include <iostream>
#include <fstream>
#include <string>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
#include <chrono>
#include <thread>

#define NUM_ANTS 4
#define WIDTH 30
#define HEIGHT 30
#define FOOD_SOURCES 3

#define CHECK_CUDA_CALL(call)                                        \
{                                                                    \
    hipError_t err = call;                                          \
    if (err != hipSuccess) {                                        \
    std::cerr << "CUDA Error: " << hipGetErrorString(err)           \
    << " in " << __FILE__ << " at line " << __LINE__ << std::endl;   \
    exit(EXIT_FAILURE);                                              \
    }                                                                \
}

void saveEnvironmentAndAnts(int* environment, Ant* ants, int width, int height, int numAnts, int iteration) {
    int* hostEnvironment = static_cast<int *>(malloc(sizeof(int) * width * height));
    Ant* hostAnts = static_cast<Ant *>(malloc(sizeof(Ant) * numAnts));

    CHECK_CUDA_CALL(hipMemcpy(hostEnvironment, environment, sizeof(int) * width * height, hipMemcpyDeviceToHost));
    CHECK_CUDA_CALL(hipMemcpy(hostAnts, ants, sizeof(Ant) * numAnts, hipMemcpyDeviceToHost));

    std::ofstream envFile("environment_" + std::to_string(iteration) + ".csv");
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            envFile << hostEnvironment[y * width + x];
            if (x < width - 1) envFile << ",";
        }
        envFile << "\n";
    }
    envFile.flush();
    envFile.close();

    std::ofstream antsFile("ants_" + std::to_string(iteration) + ".csv");
    for (int i = 0; i < numAnts; i++) {
        antsFile << hostAnts[i].x << "," << hostAnts[i].y << "," << hostAnts[i].hasFood << "\n";
    }
    antsFile.flush();
    antsFile.close();

    free(hostEnvironment);
    free(hostAnts);
}

int main() {
    int* environment;
    Ant* ants;

    hiprandState* states;

    CHECK_CUDA_CALL(hipMalloc(&environment, sizeof(int) * WIDTH * HEIGHT));
    CHECK_CUDA_CALL(hipMalloc(&ants, sizeof(Ant) * NUM_ANTS));
    CHECK_CUDA_CALL(hipMalloc(&states, sizeof(hiprandState) * NUM_ANTS));

    int* hostEnvironment = static_cast<int *>(malloc(sizeof(int) * WIDTH * HEIGHT));
    Ant* hostAnts = static_cast<Ant *>(malloc(sizeof(Ant) * NUM_ANTS));

    initializeEnvironment(hostEnvironment, WIDTH, HEIGHT, FOOD_SOURCES);
    initializeAnts(hostAnts, NUM_ANTS, WIDTH, HEIGHT);

    CHECK_CUDA_CALL(hipMemcpy(environment, hostEnvironment, sizeof(int) * WIDTH * HEIGHT, hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(ants, hostAnts, sizeof(Ant) * NUM_ANTS, hipMemcpyHostToDevice));

    free(hostEnvironment);
    free(hostAnts);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((NUM_ANTS + threadsPerBlock.x - 1) / threadsPerBlock.x);
    initializeRandomStates<<<numBlocks, threadsPerBlock>>>(states, time(0), NUM_ANTS);
    hipDeviceSynchronize();

    int iteration = 0;
    bool hasFood = true;

    while (hasFood) {
        moveAnts<<<numBlocks, threadsPerBlock>>>(ants, environment, states, NUM_ANTS, WIDTH, HEIGHT);
        hipDeviceSynchronize();

        int* checkEnvironment = static_cast<int *>(malloc(sizeof(int) * WIDTH * HEIGHT));
        CHECK_CUDA_CALL(hipMemcpy(checkEnvironment, environment, sizeof(int) * WIDTH * HEIGHT, hipMemcpyDeviceToHost));

        hasFood = false;
        for (int i = 0; i < WIDTH * HEIGHT; i++) {
            if (checkEnvironment[i] == 1) {
                hasFood = true;
                break;
            }
        }

        free(checkEnvironment);

        saveEnvironmentAndAnts(environment, ants, WIDTH, HEIGHT, NUM_ANTS, iteration);
        iteration++;
        std::cout << "Iterazione " << iteration << " completata.\n";
    }

    hipFree(environment);
    hipFree(ants);
    hipFree(states);

    return 0;
}
